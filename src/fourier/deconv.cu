#include "hip/hip_runtime.h"
/* 
Deconvlution related kernels
    1D, 2D and 3D deconvlution
    Input is FFTW format (from 0 to N/2-1 and then -N/2 to -1), flag = 0
    Output is FFTW format or CMCL-compatible mode ordering (-N/2 to N/2-1), flag = 1
legendre_rule_fast cuda version should be implemented here, g and x to constant memory
*/
#include "deconv.h"
#include "hip/hip_runtime_api.h"
#include "curafft_plan.h"


__global__ void fourier_series_appro(PCS *fseries, int N, PCS *g, PCS *x, int p){
    int idx;
    for(idx = blockDim.x * blockIdx.x + threadIdx.x; idx < N; idx+=gridDim.x*blockDim.x){
        fseries[idx] = 0;
        for(int i=0; i<p; i++){
            //why N-1 - x will change the answer
            fseries[idx] += g[i]*cos((x[i]-N+1)/(PCS)(N-1)*PI* idx);
            // if(idx==0) printf("fseries %lf\n",fseries[idx]);
        }
        fseries[idx] = 2*fseries[idx]; // add negative part
    }
}

__global__ void fourier_series_appro(PCS *fseries, PCS *k, int N, int nf, PCS *g, PCS *x, int p){
    // 2p nodes and with some error
    int idx;
    
    for(idx = blockDim.x * blockIdx.x + threadIdx.x; idx < N; idx+=gridDim.x*blockDim.x){
        fseries[idx] = 0.0;
         //printf("idx %d, k %.10lf\n", idx, k[idx]);
        
        for(int i=0; i<2*p; i++){
            
            fseries[idx] += g[i]*cos((x[i])/((PCS)(nf-1.0))*PI* k[idx]);
            //if(idx==0) printf("fseries %lf\n",fseries[idx]);

        }
        fseries[idx] = fseries[idx]; // add negative part
        //printf("%lf ",fseries[idx]);
    }
}

__global__ void fourier_series_appro(PCS *fseries, PCS *k, int N, PCS *g, PCS *x, int p){
    // 2p nodes and with some error
    int idx;
    
    for(idx = blockDim.x * blockIdx.x + threadIdx.x; idx < N; idx+=gridDim.x*blockDim.x){
        fseries[idx] = 0.0;
         //printf("idx %d, k %.10lf\n", idx, k[idx]);
        for(int i=0; i<p; i++){
            fseries[idx] += g[i]*2*cos((x[i])* k[idx]);
        }
        fseries[idx] = fseries[idx]; // add negative part
        //printf("%lf ",fseries[idx]);
    }
}

int fourier_series_appro_invoker(PCS *fseries, conv_opts opts, int N)
{
    /*
        One dimensional Fourier series approximation. f(k) = int e^{ikx} f(x) dx.
        Input: 
            opts - convolution options
            k - location of the series (on device)
            N - number of k
            flag - -1 or 1
        Output: real(fk) // on device
    */
    // comments need to be revised
    int ier = 0;
    PCS alpha = opts.kw / 2.0; // J/2, half-width of ker z-support
    // # quadr nodes in z (from 0 to J/2; reflections will be added)...
    int p = (int)(2 + 3.0 * alpha); // not sure why so large? cannot exceed MAX_NQUAD
    PCS g[MAX_NQUAD]; // intermediate result
    double x[2 * MAX_NQUAD], w[2 * MAX_NQUAD];
    legendre_compute_glr(2 * p, x, w); // only half the nodes used, eg on (0,1)
    for (int n = 0; n < p; ++n) //using 2q points testing
    {                                                              // set up nodes z_n and vals f_n
        x[n] *= alpha;                                                // rescale nodes
        PCS phi = 0.0;
        if(abs(x[n])<=opts.ES_halfwidth) phi = exp(opts.ES_beta * (sqrt(1.0 - opts.ES_c * x[n] * x[n])));
        g[n] = alpha * (PCS)w[n] * phi;  // vals & quadr wei
        // a[n] = exp(2 * PI * IMA * (PCS)(nf / 2 - z[n]) / (PCS)nf); // phase winding rates
    }
    double *d_x;
    PCS *d_g;
    checkCudaErrors(hipMalloc((void**)&d_x, sizeof(double)*p)); // change to constant memory
    checkCudaErrors(hipMalloc((void**)&d_g, sizeof(PCS)*p));

    checkCudaErrors(hipMemcpy(d_x, x, sizeof(double)*p, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_g, g, sizeof(PCS)*p, hipMemcpyHostToDevice));

    int blocksize = 512;
    fourier_series_appro<<<(N-1)/blocksize+1,blocksize>>>(fseries,N,d_g,d_x,p);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(d_g));
    checkCudaErrors(hipFree(d_x));

    return ier;
}


int fourier_series_appro_invoker(PCS *fseries, PCS *k, conv_opts opts, int N, int nf)
{
    /*
        One dimensional Fourier series approximation. f(k) = int e^{ikx} f(x) dx.
        Input: 
            opts - convolution options
            k - location of the series (on device)
            N - number of k
            nf
            flag - -1 or 1
        Output: real(fk) // on device
    */
    // comments need to be revised
    int ier = 0;
    PCS alpha = opts.kw / 2.0; // J/2, half-width of ker z-support
    printf("alpha %lf, beta %lf\n",alpha, opts.ES_beta);
    // # quadr nodes in z (from 0 to J/2; reflections will be added)...
    int p = (int)(2 + 3.0 * alpha); // not sure why so large? cannot exceed MAX_NQUAD
    PCS g[MAX_NQUAD]; // intermediate result
    double x[2 * MAX_NQUAD], w[2 * MAX_NQUAD];
    legendre_compute_glr(2 * p, x, w); 
    for (int n = 0; n < 2*p; ++n) //using 2q points testing
    {                                                              // set up nodes z_n and vals f_n
        x[n] *= alpha;                                                // rescale nodes
        PCS phi = 0.0;
        if(abs(x[n])<=opts.ES_halfwidth) phi = exp(opts.ES_beta * (sqrt(1.0 - opts.ES_c * x[n] * x[n])));
        g[n] = alpha * (PCS)w[n] * phi;  // vals & quadr wei
        // a[n] = exp(2 * PI * IMA * (PCS)(nf / 2 - z[n]) / (PCS)nf); // phase winding rates
    }
    double *d_x;
    PCS *d_g;
    checkCudaErrors(hipMalloc((void**)&d_x, sizeof(double)*2*p)); // change to constant memory
    checkCudaErrors(hipMalloc((void**)&d_g, sizeof(PCS)*2*p));

    checkCudaErrors(hipMemcpy(d_x, x, sizeof(double)*2*p, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_g, g, sizeof(PCS)*2*p, hipMemcpyHostToDevice));

    int blocksize = 512;
    fourier_series_appro<<<(N-1)/blocksize+1,blocksize>>>(fseries,k,N,nf,d_g,d_x,p);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(d_g));
    checkCudaErrors(hipFree(d_x));

    return ier;
}

int fourier_series_appro_invoker(PCS *fseries, PCS *k, conv_opts opts, int N)
{
    /*
        One dimensional Fourier series approximation. f(k) = int e^{ikx} f(x) dx.
        Input: 
            opts - convolution options
            k - location of the series (on device)
            N - number of k
            nf
            flag - -1 or 1
        Output: real(fk) // on device
    */
    // comments need to be revised
    int ier = 0;
    PCS alpha = opts.kw / 2.0; // J/2, half-width of ker z-support
#ifdef INFO
    printf("alpha %lf, beta %lf\n",alpha, opts.ES_beta);
#endif
    // # quadr nodes in z (from 0 to J/2; reflections will be added)...
    int p = (int)(2 + 3.0 * alpha); // not sure why so large? cannot exceed MAX_NQUAD
    PCS g[MAX_NQUAD]; // intermediate result
    double x[2 * MAX_NQUAD], w[2 * MAX_NQUAD];
    legendre_compute_glr(2 * p, x, w); 
    for (int n = 0; n < 2*p; ++n) //using 2q points testing
    {                                                              // set up nodes z_n and vals f_n
        x[n] *= alpha;                                                // rescale nodes
        PCS phi = 0.0;
        if(abs(x[n])<=opts.ES_halfwidth) phi = exp(opts.ES_beta * (sqrt(1.0 - opts.ES_c * x[n] * x[n])));
        g[n] = alpha * (PCS)w[n] * phi;  // vals & quadr wei
        // a[n] = exp(2 * PI * IMA * (PCS)(nf / 2 - z[n]) / (PCS)nf); // phase winding rates
    }
    double *d_x;
    PCS *d_g;
    checkCudaErrors(hipMalloc((void**)&d_x, sizeof(double)*2*p)); // change to constant memory
    checkCudaErrors(hipMalloc((void**)&d_g, sizeof(PCS)*2*p));

    checkCudaErrors(hipMemcpy(d_x, x, sizeof(double)*2*p, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_g, g, sizeof(PCS)*2*p, hipMemcpyHostToDevice));

    int blocksize = 512;
    fourier_series_appro<<<(N-1)/blocksize+1,blocksize>>>(fseries,k,N,d_g,d_x,p);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(d_g));
    checkCudaErrors(hipFree(d_x));

    return ier;
}

__global__ void deconv_1d(int N1, int nf1, CUCPX *fw, CUCPX *fk, PCS *fwkerhalf1, int flag, int type){
    /*
        One dimensional deconvlution
        N - number of modes
        nf - grid size after upsampling
        fw - fft result
        fk - final result after deconv
        fwkerhalf - half of Fourier tranform (integal) of kernel fucntion, size - N/2+1
        flag - FFTW style (other) or CMCL (1)
        type - 1 NU->U, 2 U->NU.
    */
    int idx;
    int nmodes = N1;
    int w = 0;
    int k;
    int idx_fw = 0;
    for(idx = blockIdx.x*blockDim.x + threadIdx.x; idx < nmodes; idx+=gridDim.x*blockDim.x){
        k = idx;
        if(flag == 1){
            w = k >= N1/2 ? k - N1/2 : nf1 + k - N1/2; // CMCL
        }
        else{
            w = k >= N1/2 ? nf1+k-N1 : k; // FFTW
        }
        idx_fw = w;
        if(type==1){
            fk[idx].x = fw[idx_fw].x / fwkerhalf1[abs(k-N1/2)];
            fk[idx].y = fw[idx_fw].y / fwkerhalf1[abs(k-N1/2)];
        }
        else{
            fw[idx_fw].x = fk[idx].x / fwkerhalf1[abs(k-N1/2)];
            fw[idx_fw].y = fk[idx].y / fwkerhalf1[abs(k-N1/2)];
        }
        
    }
}

__global__ void deconv_2d(int N1, int N2, int nf1, int nf2, CUCPX* fw, CUCPX* fk, PCS* fwkerhalf1,
 PCS* fwkerhalf2, int flag, int type){
     /*
        The output of cufft is from 0 to N/2-1 and then -N/2 to -1
        Should convert to -N/2 to N/2-1, then set flag = 1
     */
    int idx;
    int nmodes = N1*N2;
    int k1, k2, idx_fw, w1, w2;
    
    for(idx = blockIdx.x*blockDim.x + threadIdx.x; idx < nmodes; idx+=gridDim.x*blockDim.x){
        k1 = idx % N1;
		k2 = idx / N1;
        idx_fw = 0;
        w1 = 0;
        w2 = 0;
        if(flag == 1){
            w1 = k1 >= N1/2 ? k1-N1/2 : nf1+k1-N1/2;
		    w2 = k2 >= N2/2 ? k2-N2/2 : nf2+k2-N2/2;
        }
        else{
            w1 = k1 >= N1/2 ? nf1+k1-N1 : k1;
            w2 = k2 >= N2/2 ? nf2+k2-N2 : k2;
        }
        idx_fw = w1 + w2*nf1;
        
        
		PCS kervalue = fwkerhalf1[abs(k1-N1/2)]*fwkerhalf2[abs(k2-N2/2)];

        if(type==1){
            fk[idx].x = fw[idx_fw].x / kervalue;
            fk[idx].y = fw[idx_fw].y / kervalue;
        }
        else{
            // if(idx==(N1*N2/2+N1/2))printf("kerval %.6g..........\n",kervalue);
            fw[idx_fw].x = fk[idx].x / kervalue;
            fw[idx_fw].y = fk[idx].y / kervalue;
        }

    }
}

__global__ void deconv_3d(int N1, int N2, int N3, int nf1, int nf2, int nf3, CUCPX* fw, 
	CUCPX *fk, PCS *fwkerhalf1, PCS *fwkerhalf2, PCS *fwkerhalf3, int flag, int type)
{
    int idx;
    int nmodes = N1*N2*N3;
    int k1, k2, k3, idx_fw, w1, w2, w3;
	for(idx=blockDim.x*blockIdx.x+threadIdx.x; idx<nmodes; idx+=blockDim.x*
		gridDim.x){
		k1 = idx % N1;
		k2 = (idx / N1) % N2;
		k3 = (idx / N1 / N2);
        w1=0, w2=0, w3=0;
        idx_fw = 0;
        if(flag == 1){
            w1 = k1 >= N1/2 ? k1-N1/2 : nf1+k1-N1/2;
		    w2 = k2 >= N2/2 ? k2-N2/2 : nf2+k2-N2/2;
		    w3 = k3 >= N3/2 ? k3-N3/2 : nf3+k3-N3/2;
        }
        else{
            w1 = k1 >= N1/2 ? nf1+k1-N1 : k1;
            w2 = k2 >= N2/2 ? nf2+k2-N2 : k2;
            w3 = k3 >= N3/2 ? nf3+k3-N3 : k3;
        }
	    idx_fw = w1 + w2*nf1 + w3*nf1*nf2;

		PCS kervalue = fwkerhalf1[abs(k1-N1/2)]*fwkerhalf2[abs(k2-N2/2)]*
			fwkerhalf3[abs(k3-N3/2)];
		if(type==1){
            fk[idx].x = fw[idx_fw].x / kervalue;
            fk[idx].y = fw[idx_fw].y / kervalue;
        }
        else{
            
            fw[idx_fw].x = fk[idx].x / kervalue;
            fw[idx_fw].y = fk[idx].y / kervalue;
        }
	}
}


int curafft_deconv(curafft_plan *plan){
    /*
        invoke deconv based on dimension(s)
    */
    int ier = 0;
    int N1 = plan->ms;
    int nf1 = plan->nf1;
    int dim = plan->dim;
    int nmodes, N2, N3, nf2, nf3;
    // int batch_size = plan->batchsize;
    int flag = plan->mode_flag;
    int type = plan->type;
    int blocksize = 256;
    
    switch(dim){
        case 1:{
            nmodes = N1;
            deconv_1d<<<(nmodes-1)/blocksize+1, blocksize>>>(N1, nf1, plan->fw,plan->fk,
        plan->fwkerhalf1, flag, type);
            checkCudaErrors(hipDeviceSynchronize());
            break;
        }
        case 2:{
            N2 = plan->mt;
            nf2 = plan->nf2;
            nmodes = N1*N2;
            deconv_2d<<<(nmodes-1)/blocksize+1, blocksize>>>(N1, N2, nf1, nf2, plan->fw,plan->fk,
        plan->fwkerhalf1, plan->fwkerhalf2, flag, type);
            checkCudaErrors(hipDeviceSynchronize());
            break;
        }
        case 3:{
            N2 = plan->mt;
            N3 = plan->mu;
            nf2 = plan->nf2;
            nf3 = plan->nf3;
            nmodes = N1*N2*N3;
            deconv_3d<<<(nmodes-1)/blocksize+1, blocksize>>>(N1, N2, N3, nf1, nf2, nf3, plan->fw,plan->fk,
        plan->fwkerhalf1, plan->fwkerhalf2, plan->fwkerhalf3, flag, type);
            checkCudaErrors(hipDeviceSynchronize());
            break;
        }
        default:{
            ier = 1; //error
        }
    }

    return ier;
}


//------------------Below this line, the content is just for Radio Astronomy---------------------

__global__ void w_term_deconv(int N1, int N2, CUCPX* fk, PCS* fwkerhalf, PCS i_center, PCS o_center ,PCS xpixelsize, PCS ypixelsize, int flag){
    /*
        w term deconvolution
        Due to the symetric property, just calculate (N1/2+1)*(N2/2+1), input and output are CMCL format
        Parameters:
            N1 and N2 are image size
            fk - the result after ft
            fwkerhalf - correction factor
            i|o_center - input or output center
            pixelsize - degrees per pixel
            flag
    */
    // 
    int idx;
    int nmodes = N1*N2; 
    int idx_fw = 0;
    for(idx = blockIdx.x*blockDim.x + threadIdx.x; idx < nmodes; idx+=gridDim.x*blockDim.x){
        int row = idx / N1;
        int col = idx % N1;
        PCS phase = ((sqrt(1.0 - pow((row-N2/2)*xpixelsize,2) - pow((col-N1/2)*ypixelsize,2)) - 1)-o_center)*i_center*flag; // caused by shifting ({i*(u+u_c)*x_c})

        idx_fw = abs(col-N1/2)+abs(row-N2/2)*(N1/2+1);
        // if(idx==0)printf("gpu fwkerhalf %.10g, exp .real %.10g sin %.10g,  %.10g\n",fwkerhalf[idx_fw], cos(phase), sin(phase), (fk[idx].x*cos(phase)-fk[idx].y*sin(phase)) / fwkerhalf[idx_fw]);
        CUCPX temp;
        temp.x = (fk[idx].x*cos(phase)-fk[idx].y*sin(phase)) / fwkerhalf[idx_fw];
        temp.y = (fk[idx].x*sin(phase)+fk[idx].y*cos(phase))  / fwkerhalf[idx_fw];
        fk[idx] = temp;
    }
}



int curadft_w_deconv(curafft_plan *plan, PCS xpixelsize, PCS ypixelsize){
    /*
        w term deconvolution invoker
    */
    int ier = 0;
    int blocksize = 512;
    int N = plan->ms*plan->mt;
    PCS i_center = plan->ta.i_center[0];
    PCS o_center = plan->ta.o_center[0];
    int flag = plan->iflag;
    w_term_deconv<<<(N-1)/blocksize+1,blocksize>>>(plan->ms,plan->mt,plan->fk,plan->fwkerhalf3,i_center,o_center,xpixelsize,ypixelsize,flag);
    checkCudaErrors(hipDeviceSynchronize());
    
    
    return ier;
}