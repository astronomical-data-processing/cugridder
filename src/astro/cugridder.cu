#include "hip/hip_runtime.h"
/* --------cugridder-----------
    1. gridder_setting
        fov and other astro related setting
        opt setting
        plan setting
        bin setting
    2. gridder_execution
    3. gridder_destroy
*/

#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <complex>
#include <hipfft/hipfft.h>
#include <fstream>
#include "conv_interp_invoker.h"
#include "ragridder_plan.h"
#include "curafft_plan.h"
#include "cuft.h"
#include "precomp.h"
#include "ra_exec.h"
#include "utils.h"
#include "hipfft/hipfft.h"
#include "deconv.h"
#include "cugridder.h"

int setup_gridder_plan(int N1, int N2, PCS fov, int lshift, int mshift, int nrow, PCS *d_w, CUCPX *d_c, conv_opts copts, ragridder_plan *gridder_plan, curafft_plan *plan)
{
    /*
    gridder related parameters setting
    Input: 
        N1, N2 - image size
        fov - field of view
        nrow - number of coordinate
        d_w - w array on device
        d_c - vis value
    */
    gridder_plan->fov = fov;
    gridder_plan->width = N1;
    gridder_plan->height = N2;
    gridder_plan->nrow = nrow;
    // determain number of w
    // ignore shift

    // degree per pixel
    gridder_plan->pixelsize_x = fov / 180.0 * PI / (PCS)N2;
    gridder_plan->pixelsize_y = fov / 180.0 * PI / (PCS)N1;
    PCS xpixelsize = gridder_plan->pixelsize_x;
    PCS ypixelsize = gridder_plan->pixelsize_y;
    PCS l_min = lshift - 0.5 * xpixelsize * N2;
    PCS l_max = l_min + xpixelsize * (N2 - 1);

    PCS m_min = mshift - 0.5 * ypixelsize * N1;
    PCS m_max = m_min + ypixelsize * (N1 - 1);

    //double upsampling_fac = copts.upsampfac;
    PCS n_lm = sqrt(1.0 - pow(l_min, 2) - pow(m_min, 2));
    
    // nshift = (no_nshift||(!do_wgridding)) ? 0. : -0.5*(nm1max+nm1min);

    // get max min of input and output
    PCS i_max, i_min;
    PCS o_min;
    get_max_min(i_max, i_min, d_w, gridder_plan->nrow);
    plan->ta.i_center[0] = (i_max + i_min) / (PCS)2.0;
    plan->ta.i_half_width[0] = (i_max - i_min) / (PCS)2.0;

    o_min = n_lm-1;
    plan->ta.o_center[0] =  o_min / (PCS)2.0;
    plan->ta.o_half_width[0] = abs(o_min / (PCS)2.0);

    
    // get number of w planes, scaling ratio gamma
    set_nhg_type3(plan->ta.o_half_width[0], plan->ta.i_half_width[0], plan->copts, plan->nf1, plan->ta.h[0], plan->ta.gamma[0]); //temporately use nf1
#ifdef INFO
    printf("U_width %lf, U_center %lf, X_width %.10lf, X_center %.10lf, gamma %lf, nf %d, h %lf\n",
           plan->ta.i_half_width[0], plan->ta.i_center[0], plan->ta.o_half_width[0], plan->ta.o_center[0], plan->ta.gamma[0], plan->nf1, plan->ta.h[0]);
#endif
    // to cura_prestage
    // wgt * vis
    // if(gridder_plan->kv.weight!=NULL&&plan->copts.direction==1){
    //     PCS *d_wgt;
    //     checkCudaErrors(hipMalloc((void**)&d_wgt,sizeof(PCS)*nrow));
    //     checkCudaErrors(hipMemcpy(d_wgt,gridder_plan->kv.weight,sizeof(PCS)*nrow,hipMemcpyHostToDevice));

    //     matrix_elementwise_multiply_invoker(d_c,d_wgt,nrow);
    //     checkCudaErrors(hipFree(d_wgt)); // to save memory
    // }
    // // u_j to u_j' x_k to x_k' c_j to c_j'
    // checkCudaErrors(hipMalloc((void **)&plan->d_x, sizeof(PCS) * (N1 / 2 + 1) * (N2 / 2 + 1)));
    // w_term_k_generation(plan->d_x, N1, N2, gridder_plan->pixelsize_x, gridder_plan->pixelsize_y);

    // pre_stage_invoker(plan->ta.i_center, plan->ta.o_center, plan->ta.gamma, plan->ta.h, d_w, NULL, NULL, plan->d_x, NULL, NULL, d_c, gridder_plan->nrow,(N1 / 2 + 1) * (N2 / 2 + 1), 1, 1, plan->iflag);
    
    
    gridder_plan->num_w = plan->nf1;

    return 0;
}

// the bin sort should be completed at gridder_settting

int gridder_setting(int N1, int N2, int method, int kerevalmeth, int w_term_method, PCS tol, int direction, double sigma, int iflag,
                    int batchsize, int M, int channel, PCS fov, visibility *pointer_v, PCS *d_u, PCS *d_v, PCS *d_w,
                    CUCPX *d_c, curafft_plan *plan, ragridder_plan *gridder_plan)
{
    /*
        N1, N2 - number of Fouier modes
        method - gridding method
        kerevalmeth - gridding kernel evaluation method
        tol - tolerance (epsilon)
        direction - 1 vis to image, 0 image to vis
        sigma - upsampling factor
        iflag - flag for fourier transform
        batchsize - number of batch in  cufft (used for handling piece by piece)
        M - number of nputs (visibility)
        channel - number of channels
        wgt - weight
        freq - frequency
        d_u, d_v, d_w - wavelengths in different dimensions, x is on host, d_x is on device
        d_c - value of visibility
        ****issue, degridding
    */
    int ier = 0;

    // fov and other astro related setting +++

    // get effective coordinates: *1/lambda
    PCS f_over_c = pointer_v->frequency[0]/SPEEDOFLIGHT;
    // printf("foverc %lf\n",f_over_c);
   
    int sign;
    sign = pointer_v->sign;
    get_effective_coordinate_invoker(d_u,d_v,d_w,f_over_c,pointer_v->pirange,M,sign);

    // PCS *w = (PCS *) malloc(sizeof(PCS)*M);
    // checkCudaErrors(hipMemcpy(w,d_w,sizeof(PCS)*M,hipMemcpyDeviceToHost));
   
    // opts and copts setting
    plan->opts.gpu_device_id = 0;
    plan->opts.upsampfac = sigma;
    plan->opts.gpu_sort = 0;
    plan->opts.gpu_binsizex = -1;
    plan->opts.gpu_binsizey = -1;
    plan->opts.gpu_binsizez = -1;
    plan->opts.gpu_kerevalmeth = kerevalmeth;
    plan->opts.gpu_conv_only = 0;
    plan->opts.gpu_gridder_method = method;

    
    ier = setup_conv_opts(plan->copts, tol, sigma, 1, direction, kerevalmeth); //check the arguements pirange = 1
    
    int fftsign = (direction > 0) ? 1 : -1;
    plan->iflag = fftsign; 

    if (fftsign==1) plan->type = 1;
    else plan->type = 2; // will be used at deconv

    if (ier != 0)
        printf("setup_error\n");
    
    // gridder plan setting
    // cuda stream malloc in setup_plan
    gridder_plan->channel = channel;
    gridder_plan->w_term_method = w_term_method;
    gridder_plan->speedoflight = SPEEDOFLIGHT;
    gridder_plan->kv.u = pointer_v->u;
    gridder_plan->kv.v = pointer_v->v;
    gridder_plan->kv.w = pointer_v->w;
    gridder_plan->kv.vis = pointer_v->vis;
    gridder_plan->kv.weight = pointer_v->weight;
    gridder_plan->kv.frequency = pointer_v->frequency;
    gridder_plan->kv.pirange = pointer_v->pirange;

    
    setup_gridder_plan(N1, N2, fov, 0, 0, M, d_w, d_c, plan->copts, gridder_plan, plan);

    int nf1 = get_num_cells(N1, plan->copts);
    int nf2 = get_num_cells(N2, plan->copts);
    int nf3 = gridder_plan->num_w;
    if (w_term_method)
        plan->dim = 3;
    else
        plan->dim = 2;
    
    setup_plan(nf1, nf2, nf3, M, d_v, d_u, d_w, d_c, plan);
    // printf("input data checking cugridder...\n");
    //         PCS *temp = (PCS*)malloc(sizeof(PCS)*10);
    //         printf("u v w and vis\n");
    //         hipMemcpy(temp,d_u,sizeof(PCS)*10,hipMemcpyDeviceToHost);
    //         for(int i=0;i<10;i++)
    //         printf("%.3lf ",temp[i]);
    //         printf("\n");

    plan->ms = N1;
    plan->mt = N2;
    plan->mu = 1;
    plan->execute_flow = 1;
    //plan->fw = NULL; 
    batchsize = gridder_plan->num_w;

    // plan->copts.direction = direction; // 1 inverse, 0 forward

    // fourier_series_appro_invoker(plan->fwkerhalf1, plan->copts, plan->nf1 / 2 + 1);
    // fourier_series_appro_invoker(plan->fwkerhalf2, plan->copts, plan->nf2 / 2 + 1);

    // if (w_term_method)
    // {
    //     // improved_ws
    //     checkCudaErrors(hipFree(plan->fwkerhalf3));
    //     checkCudaErrors(hipMalloc((void **)&plan->fwkerhalf3, sizeof(PCS) * (N1 / 2 + 1) * (N2 / 2 + 1)));
        
    //     fourier_series_appro_invoker(plan->fwkerhalf3, plan->d_x, plan->copts, (N1 / 2 + 1) * (N2 / 2 + 1)); // correction with k, may be wrong, k will be free in this function
    // }

    // PCS *fwkerhalf1 = (PCS *)malloc(sizeof(PCS) * (plan->nf1 / 2 + 1));
    // PCS *fwkerhalf2 = (PCS *)malloc(sizeof(PCS) * (plan->nf2 / 2 + 1));

    // hipMemcpy(fwkerhalf1, plan->fwkerhalf1, sizeof(PCS) * (plan->nf1 / 2 + 1), hipMemcpyDeviceToHost);
    // hipMemcpy(fwkerhalf2, plan->fwkerhalf2, sizeof(PCS) * (plan->nf2 / 2 + 1), hipMemcpyDeviceToHost);

    // cufft plan setting
    hipfftHandle fftplan;
    int n[] = {plan->nf2, plan->nf1};
    int inembed[] = {plan->nf2, plan->nf1};
    int onembed[] = {plan->nf2, plan->nf1};
    
    if(MAX_CUFFT_ELEM/plan->nf1/plan->nf2<plan->nf3){
        batchsize = MAX_CUFFT_ELEM/plan->nf1/plan->nf2;
        hipfftHandle fftplanl;
        int remain_batch = plan->nf3%batchsize;
        hipfftPlanMany(&fftplanl, 2, n, inembed, 1, inembed[0] * inembed[1],
                  onembed, 1, onembed[0] * onembed[1], CUFFT_TYPE, remain_batch);
        plan->fftplan_l = fftplanl;
    }
    // check, multi cufft for different w ??? how to set
    // hipfftCreate(&fftplan);
    // hipfftPlan2d(&fftplan,n[0],n[1],CUFFT_TYPE);
    // the bach size sets as the num of w when memory is sufficent. Alternative way, set as a smaller number when memory is insufficient.
    // and handle this piece by piece
    hipfftPlanMany(&fftplan, 2, n, inembed, 1, inembed[0] * inembed[1],
                  onembed, 1, onembed[0] * onembed[1], CUFFT_TYPE, batchsize); //There's a hard limit of roughly 2^27 elements in a plan!!!!!!!!!
    plan->fftplan = fftplan;
    plan->batchsize = batchsize;
    
    // u and v scaling *pixelsize
    rescaling_real_invoker(d_u,gridder_plan->pixelsize_x,gridder_plan->nrow);
    rescaling_real_invoker(d_v,gridder_plan->pixelsize_y,gridder_plan->nrow);
    
    // fw malloc
    // printf("nf1, nf2, nf3: (%d,%d,%d) %d\n",plan->nf1,plan->nf2,plan->nf3,plan->nf1*plan->nf2*plan->nf3);
#ifdef INFO
    show_mem_usage();
    printf("nf1, nf2, nf3: (%d,%d,%d) %d\n",plan->nf1,plan->nf2,plan->nf3,plan->nf1*plan->nf2*plan->nf3);
#endif
    
    return ier;
}

int gridder_execution(curafft_plan *plan, ragridder_plan *gridder_plan)
{
    /*
    Execute conv, fft, dft, correction
    */
    int ier = 0;
    // Mult-GPU support: set the CUDA Device ID:
    // int orig_gpu_device_id;
    // hipGetDevice(& orig_gpu_device_id);
    // hipSetDevice(d_plan->opts.gpu_device_id);
    int direction = plan->copts.direction;

    if (direction == 1)
    {
        ier = exec_vis2dirty(plan, gridder_plan);
    }
    else
    {
        // forward not implement yet
        ier = exec_dirty2vis(plan, gridder_plan);
    }

    // Multi-GPU support: reset the device ID
    // hipSetDevice(orig_gpu_device_id);
    return ier;
}

int gridder_destroy(curafft_plan *plan, ragridder_plan *gridder_plan)
{
    // free memory
    int ier = 0;
    checkCudaErrors(hipFree(plan->d_x));
    curafft_free(plan);
    //checkCudaErrors(hipDeviceReset());
    free(plan);
    free(gridder_plan->dirty_image);
    free(gridder_plan->kv.u);
    free(gridder_plan->kv.v);
    free(gridder_plan->kv.w);
    free(gridder_plan->kv.vis);
    free(gridder_plan->kv.frequency);
    free(gridder_plan->kv.weight);
    // free(gridder_plan->kv.flag);
    free(gridder_plan);
    return ier;
}

int py_gridder_destroy(curafft_plan *plan, ragridder_plan *gridder_plan)
{   
    // free memory
    int ier=0;
    if (plan->opts.gpu_sort)
    {
        checkCudaErrors(hipFree(plan->cell_loc));
    }
    hipfftDestroy(plan->fftplan);
    checkCudaErrors(hipFree(plan->d_x));
    checkCudaErrors(hipFree(plan->fwkerhalf3));
    checkCudaErrors(hipFree(plan->fwkerhalf2));
    checkCudaErrors(hipFree(plan->fwkerhalf1));
    checkCudaErrors(hipFree(plan->d_c));
    checkCudaErrors(hipFree(plan->fw));
    checkCudaErrors(hipFree(plan->fk));
    free(plan);
    free(gridder_plan);
    return ier;
}
// -------------gridder warpper-----------------
int ms2dirty_exec(int nrow, int nxdirty, int nydirty, PCS fov, PCS freq, PCS *uvw,
             CPX *vis, PCS *wgt,  CPX *dirty, PCS epsilon, PCS sigma, int sign){
    /*
    generating image from ms(vis)
    Input:
        nrow - number of coordinates
        nxdirty nydirty - image size
        fov - field of view
        freq - freqency
        uvw - coordinate [nrow,3]
        vis - visibility
        epsilon - expected error
        sigma - upsampling factor
    Output:
        d_dirty - dirty image on device
    */
    int ier = 0;
    //checkCudaErrors(hipSetDevice(0));
#ifdef TIME
    hipEvent_t start, stop;
	float milliseconds = 0;
	float totaltime = 0;
    float copytime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    hipEventRecord(start);
#endif
    //------------transpose uvw------------

    PCS *d_uvw;
    CUCPX *d_vis;
    checkCudaErrors(hipMalloc((void**)&d_uvw, 3 * nrow * sizeof(PCS)));
    checkCudaErrors(hipMemcpy(d_uvw,uvw,3 * nrow * sizeof(PCS),hipMemcpyHostToDevice));
    matrix_transpose_invoker(d_uvw,3,nrow); // will use a temp arr with same size as uvw
    checkCudaErrors(hipMalloc((void**)&d_vis, nrow * sizeof(CUCPX)));
    checkCudaErrors(hipMemcpy(d_vis,  vis, nrow * sizeof(CUCPX), hipMemcpyHostToDevice));
    
    //------------device memory malloc------------
    PCS *d_u, *d_v, *d_w;
    d_u = d_uvw;
    d_v = d_uvw+nrow;
    d_w = d_uvw+2*nrow;

    PCS *f_over_c = (PCS*) malloc(sizeof(PCS));
    f_over_c[0] = freq / SPEEDOFLIGHT;

    /* -------------- cugridder-----------------*/
	// plan setting
	curafft_plan *plan;

	ragridder_plan *gridder_plan;

	plan = new curafft_plan();
    gridder_plan = new ragridder_plan();
    memset(plan, 0, sizeof(curafft_plan));
    memset(gridder_plan, 0, sizeof(ragridder_plan));
	
	visibility *pointer_v;
	pointer_v = (visibility *)malloc(sizeof(visibility));
	pointer_v->u = uvw;
	pointer_v->v = uvw+nrow;
	pointer_v->w = uvw+2*nrow; //wrong
	pointer_v->vis = vis;
	pointer_v->frequency = &freq;
	pointer_v->weight = wgt;
	pointer_v->pirange = 0;
    pointer_v->sign = sign;
	int direction = 1; //vis to image
    //---------STEP1: gridder setting---------------
    ier = gridder_setting(nydirty,nxdirty,0,0,1,epsilon,direction,sigma,0,1,nrow,1,fov,pointer_v,d_u,d_v,d_w,d_vis
		,plan,gridder_plan);
    //print the setting result
	free(pointer_v);
	if(ier == 1){
		printf("errors in gridder setting\n");
		return ier;
	}
    CUCPX *d_dirty;
    checkCudaErrors(hipMalloc((void**)&d_dirty,sizeof(CUCPX)*nxdirty*nydirty));
    plan->fk = d_dirty;
#ifdef TIME
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] Setting time:\t\t %.3g s\n", (milliseconds)/1000);
#endif

    //---------STEP2: gridder execution---------------
#ifdef TIME
    hipEventRecord(start);
#endif
    ier = gridder_execution(plan,gridder_plan);
#ifdef TIME
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] Exec time:\t\t %.3g s\n", milliseconds/1000);
    hipEventRecord(start);
#endif
    checkCudaErrors(hipMemcpy(dirty,plan->fk,sizeof(CUCPX)*nxdirty*nydirty,hipMemcpyDeviceToHost));
	if(ier == 1){
		printf("errors in gridder execution\n");
		return ier;
	}
    //---------STEP3: gridder destroy-----------------
    checkCudaErrors(hipFree(d_uvw));
    ier = py_gridder_destroy(plan,gridder_plan);
	if(ier == 1){
		printf("errors in gridder destroy\n");
		return ier;
	}
#ifdef TIME
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] Result copy and detroy time:\t\t %.3g s\n", milliseconds/1000);
	printf("[time  ] Total time:\t\t %.3g s\n", totaltime/1000);
#endif
    //checkCudaErrors(hipDeviceReset());
    return ier;
}

int dirty2ms_exec(int nrow, int nxdirty, int nydirty, PCS fov, PCS freq, PCS *uvw,
             CPX *vis, PCS *wgt,  CPX *dirty, PCS epsilon, PCS sigma, int sign){
    // +++ c dirty c/*.
    /*
    from image to ms(vis)
    Input:
        nrow - number of coordinates
        nxdirty nydirty - image size
        fov - field of view
        freq - freqency
        uvw - coordinate [nrow,3]
        vis - visibility
        epsilon - expected error
        sigma - upsampling factor
    Output:
        d_dirty - dirty image on device
    */
    int ier = 0;
    //checkCudaErrors(hipSetDevice(0));
#ifdef TIME
    hipEvent_t start, stop;
	float milliseconds = 0;
	float totaltime = 0;
    float copytime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    hipEventRecord(start);
#endif
    //------------transpose uvw------------

    PCS *d_uvw;
    CUCPX *d_vis;
    checkCudaErrors(hipMalloc((void**)&d_uvw, 3 * nrow * sizeof(PCS)));
    checkCudaErrors(hipMemcpy(d_uvw,uvw,3 * nrow * sizeof(PCS),hipMemcpyHostToDevice));
    matrix_transpose_invoker(d_uvw,3,nrow); // will use a temp arr with same size as uvw
    
    CUCPX *d_dirty;
    checkCudaErrors(hipMalloc((void**)&d_dirty,sizeof(CUCPX)*nxdirty*nydirty));
    checkCudaErrors(hipMemcpy(d_dirty,dirty,sizeof(CUCPX)*nxdirty*nydirty,hipMemcpyHostToDevice));
    //------------device memory malloc------------
    PCS *d_u, *d_v, *d_w;
    d_u = d_uvw;
    d_v = d_uvw+nrow;
    d_w = d_uvw+2*nrow;

    PCS *f_over_c = (PCS*) malloc(sizeof(PCS));
    f_over_c[0] = freq / SPEEDOFLIGHT;

    /* -------------- cugridder-----------------*/
	// plan setting
	curafft_plan *plan;

	ragridder_plan *gridder_plan;

	plan = new curafft_plan();
    gridder_plan = new ragridder_plan();
    memset(plan, 0, sizeof(curafft_plan));
    memset(gridder_plan, 0, sizeof(ragridder_plan));
	
	visibility *pointer_v;
	pointer_v = (visibility *)malloc(sizeof(visibility));
	pointer_v->u = uvw;
	pointer_v->v = uvw+nrow;
	pointer_v->w = uvw+2*nrow; //wrong
	pointer_v->vis = vis;
	pointer_v->frequency = &freq;
	pointer_v->weight = wgt;
	pointer_v->pirange = 0;
    pointer_v->sign = sign;

    plan->fk = d_dirty;
	int direction = 0; 
    //---------STEP1: gridder setting---------------
    ier = gridder_setting(nydirty,nxdirty,0,0,1,epsilon,direction,sigma,-1,1,nrow,1,fov,pointer_v,d_u,d_v,d_w,NULL
		,plan,gridder_plan);
	free(pointer_v);
	if(ier == 1){
		printf("errors in gridder setting\n");
		return ier;
	}

    checkCudaErrors(hipMalloc((void**)&d_vis, nrow * sizeof(CUCPX)));
    checkCudaErrors(hipMemset(d_vis,0,nrow * sizeof(CUCPX)));
    plan->d_c = d_vis;
#ifdef TIME
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] Setting time:\t\t %.3g s\n", (milliseconds)/1000);
#endif

    //---------STEP2: gridder execution---------------
#ifdef TIME
    hipEventRecord(start);
#endif
    ier = gridder_execution(plan,gridder_plan);
#ifdef TIME
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] Exec time:\t\t %.3g s\n", milliseconds/1000);
    hipEventRecord(start);
#endif
    checkCudaErrors(hipMemcpy(vis,d_vis,sizeof(CUCPX)*nrow,hipMemcpyDeviceToHost));
	if(ier == 1){
		printf("errors in gridder execution\n");
		return ier;
	}
    //---------STEP3: gridder destroy-----------------
    checkCudaErrors(hipFree(d_uvw));
    ier = py_gridder_destroy(plan,gridder_plan);
	if(ier == 1){
		printf("errors in gridder destroy\n");
		return ier;
	}
#ifdef TIME
    hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	totaltime += milliseconds;
	printf("[time  ] Result copy and detroy time:\t\t %.3g s\n", milliseconds/1000);
	printf("[time  ] Total time:\t\t %.3g s\n", totaltime/1000);
#endif
    //checkCudaErrors(hipDeviceReset());
    return ier;
}



// a litter bit messy, not know how to handle as one function when wgt can be None or not in python
int ms2dirty_2(int nrow, int nxdirty, int nydirty, PCS fov, PCS freq, PCS *uvw,
             CPX *vis, PCS *wgt, CPX *dirty, PCS epsilon, PCS sigma, int sign){
    int ier = 0;
    ier = ms2dirty_exec(nrow,nxdirty,nydirty,fov,freq,uvw,vis,wgt,dirty,epsilon,sigma,sign);
    return ier;
}

int ms2dirty_1(int nrow, int nxdirty, int nydirty, PCS fov, PCS freq, PCS *uvw,
             CPX *vis, CPX *dirty, PCS epsilon, PCS sigma, int sign){
    int ier = 0;
    ier = ms2dirty_exec(nrow,nxdirty,nydirty,fov,freq,uvw,vis,NULL,dirty,epsilon,sigma,sign);
    return ier;
}

int dirty2ms_1(int nrow, int nxdirty, int nydirty, PCS fov, PCS freq, PCS *uvw,
             CPX *vis, CPX *dirty, PCS epsilon, PCS sigma, int sign){
    int ier = 0;
    ier = dirty2ms_exec(nrow,nxdirty,nydirty,fov,freq,uvw,vis,NULL,dirty,epsilon,sigma,sign);
    return ier;
}

int dirty2ms_2(int nrow, int nxdirty, int nydirty, PCS fov, PCS freq, PCS *uvw,
             CPX *vis, PCS *wgt, CPX *dirty, PCS epsilon, PCS sigma, int sign){
    int ier = 0;
    ier = dirty2ms_exec(nrow,nxdirty,nydirty,fov,freq,uvw,vis,wgt,dirty,epsilon,sigma,sign);
    return ier;
}