#include <iostream>
#include <iomanip>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <thrust/complex.h>
#include <algorithm>
//#include <thrust>
using namespace thrust;

#include "conv_interp_invoker.h"

#include "cuft.h"
#include "utils.h"

///conv improved WS, method 0 correctness cheak

int main(int argc, char *argv[])
{

	//gpu_method == 0, nupts driven
	int N1, N2, N3;
	PCS sigma = 2.0;
	int M; // input
	if (argc < 4)
	{
		fprintf(stderr,
				"Usage: conv3d method nupts_distr nf1 nf2 nf3 [maxsubprobsize [M [tol [kerevalmeth [sort]]]]]\n"
				"Arguments:\n"
				"  method: One of\n"
				"    0: nupts driven,\n"
				"    2: sub-problem, or\n"
				"  N1, N2 : image size.\n"
				"  M: The number of non-uniform points.\n"
				"  tol: NUFFT tolerance (default 1e-6).\n"
				"  kerevalmeth: Kernel evaluation method; one of\n"
				"     0: Exponential of square root (default), or\n"
				"     1: Horner evaluation.\n");
		return 1;
	}
	//no result
	double w;
	int method;
	sscanf(argv[1], "%d", &method);
	sscanf(argv[2], "%lf", &w);
	N1 = (int)w; // so can read 1e6 right!
	sscanf(argv[3], "%lf", &w);
	N2 = (int)w; // so can read 1e6 right!
	sscanf(argv[4], "%lf", &w);
	N3 = (int)w;
	M = N1 * N2 * N3;
	if (argc > 5)
	{
		sscanf(argv[5], "%lf", &w);
		M = (int)w; // so can read 1e6 right!
	}

	PCS tol = 1e-10;
	if (argc > 6)
	{
		sscanf(argv[6], "%lf", &w);
		tol = (PCS)w; // so can read 1e6 right!
	}

	int kerevalmeth = 0;
	if (argc > 7)
	{
		sscanf(argv[7], "%d", &kerevalmeth);
	}

	// fov and 1 pixel corresonding to pix_deg degree

	N1 = 5;
	N2 = 5;
	N3 = 2;
	M = 50; //for correctness checking
	//int ier;
	PCS *x, *y, *z;
	CPX *c, *fw;
	x = (PCS *)malloc(M * sizeof(PCS)); //Allocates page-locked memory on the host.
	y = (PCS *)malloc(M * sizeof(PCS));
	z = (PCS *)malloc(M * sizeof(PCS));
	c = (CPX *)malloc(M * sizeof(CPX));

	//hipHostMalloc(&fw,nf1*nf2*nf3*sizeof(CPX)); //malloc after plan setting

	PCS *d_x, *d_y, *d_z;
	CUCPX *d_c, *d_fw;
	checkCudaErrors(hipMalloc(&d_x, M * sizeof(PCS)));
	checkCudaErrors(hipMalloc(&d_y, M * sizeof(PCS)));
	checkCudaErrors(hipMalloc(&d_z, M * sizeof(PCS)));
	checkCudaErrors(hipMalloc(&d_c, M * sizeof(CUCPX)));
	//checkCudaErrors(hipMalloc(&d_fw,8*nf1*nf2*nf1*sizeof(CUCPX)));

	//generating data
	int nupts_distribute = 0;
	switch (nupts_distribute)
	{
	case 0: //uniform
	{
		for (int i = 0; i < M; i++)
		{
			x[i] = M_PI * randm11();
			y[i] = M_PI * randm11();
			z[i] = M_PI * randm11();
			c[i].real(1.0); //back to random11()
			c[i].imag(1.0);
		}
	}
	break;
	case 1: // concentrate on a small region
	{
		for (int i = 0; i < M; i++)
		{
			x[i] = M_PI * rand01() / N1 * 16;
			y[i] = M_PI * rand01() / N2 * 16;
			z[i] = M_PI * rand01() / N2 * 16;
			c[i].real(randm11());
			c[i].imag(randm11());
		}
	}
	break;
	default:
		std::cerr << "not valid nupts distr" << std::endl;
		return 1;
	}
	double a[5] = {-PI / 2, -PI / 3, 0, PI / 3, PI / 2};
	for (int i = 0; i < 50; i++)
	{
		x[i] = a[i / 10];
		y[i] = a[i % 5];
		z[i] = a[i / 25 * 4];
	}

	//printf("generated data, x[1] %2.2g, y[1] %2.2g , z[1] %2.2g, c[1] %2.2g\n",x[1] , y[1], z[1], c[1].real());
	//data transfer
	checkCudaErrors(hipMemcpy(d_x, x, M * sizeof(PCS), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_y, y, M * sizeof(PCS), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_z, z, M * sizeof(PCS), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_c, c, M * sizeof(CUCPX), hipMemcpyHostToDevice));

	curafft_plan *h_plan = new curafft_plan();
	memset(h_plan, 0, sizeof(curafft_plan));

	// opts and copts setting
	h_plan->opts.gpu_conv_only = 1;
	h_plan->opts.gpu_gridder_method = method;
	h_plan->opts.gpu_kerevalmeth = kerevalmeth;
	h_plan->opts.gpu_sort = 1;
	h_plan->opts.upsampfac = sigma;
	h_plan->dim = 3;
	// h_plan->copts.pirange = 1;
	// some plan setting
	// h_plan->w_term_method = w_term_method;

	int ier = setup_conv_opts(h_plan->copts, tol, sigma, 1, 1, kerevalmeth); //check the arguements

	if (ier != 0)
		printf("setup_error\n");

	// plan setting
	int nf1 = (int)N1 * sigma;
	int nf2 = (int)N2 * sigma;
	int nf3 = (int)N3 * sigma;
	ier = setup_plan(nf1, nf2, nf3, M, d_x, d_y, d_z, d_c, h_plan); //cautious the number of plane using N1 N2 to get nf1 nf2

	// printf("the kw is %d\n", h_plan->copts.kw);
	int f_size = nf1 * nf2 * nf3;
	fw = (CPX *)malloc(sizeof(CPX) * f_size);
	checkCudaErrors(hipMalloc(&d_fw, f_size * sizeof(CUCPX)));

	h_plan->fw = d_fw;
	//checkCudaErrors(hipHostMalloc(&fw,nf1*nf2*h_plan->num_w*sizeof(CPX))); //malloc after plan setting
	//checkCudaErrors(hipMalloc( &d_fw,( nf1*nf2*(h_plan->num_w)*sizeof(CUCPX) ) ) ); //check

	std::cout << std::scientific << std::setprecision(3); //setprecision not define

	hipEvent_t cuda_start, cuda_end;

	float kernel_time;

	hipEventCreate(&cuda_start);
	hipEventCreate(&cuda_end);

	hipEventRecord(cuda_start);

	// convolution
	curafft_conv(h_plan); //add to include
	hipEventRecord(cuda_end);

	hipEventSynchronize(cuda_start);
	hipEventSynchronize(cuda_end);

	hipEventElapsedTime(&kernel_time, cuda_start, cuda_end);

	// checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(fw, d_fw, sizeof(CUCPX) * f_size, hipMemcpyDeviceToHost));

	//int nf3 = h_plan->num_w;
	printf("Method %d (nupt driven) %d NU pts to #%d U pts in %.3g s\n",
		   h_plan->opts.gpu_gridder_method, M, nf1 * nf2 * nf3, kernel_time / 1000);

	curafft_free(h_plan);

	std::cout << "[result-input]" << std::endl;
	for (int k = 0; k < nf3; k++)
	{
		for (int j = 0; j < nf2; j++)
		{
			for (int i = 0; i < nf1; i++)
			{
				printf(" (%2.3g,%2.3g)", fw[i + j * nf1 + k * nf2 * nf1].real(),
					   fw[i + j * nf1 + k * nf2 * nf1].imag());
			}
			std::cout << std::endl;
		}
		std::cout << std::endl;
	}
	std::cout << "----------------------------------------------------------------" << std::endl;

	checkCudaErrors(hipDeviceReset());
	free(x);
	free(y);
	free(z);
	free(c);
	free(fw);

	return 0;
}